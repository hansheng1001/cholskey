// #include <cublas_v2.h>

#include <hip/hip_runtime.h>
#include <iostream>
#include <hipsolver.h>
using namespace std;


// void tc_rpotrf(cublasHandle_t cublas_handle, cusolverDnHandle_t cusolver_handle, int n, float *A, int lda, float *work, __half *hwork, int nb, int *devInfo)
// {
//     if(n <= nb)
//     {
//         cusolverDnSpotrf(cusolver_handle, CUBLAS_FILL_MODE_LOWER,
//                         nb, A, lda, work, lwork, devInfo);
//         return;
//     }


//     tc_rpotrf(cublas_handle, cusolver_handle, n/2, A, lda, work, hwork, nb, devInfo);

//     tc_rtrsm(cublas_handle, n/2, n/2, A, lda, A+n/2, lda, hwork, trsm_nb);
    
//     tc_syrk(cublas_handle, n/2, n/2, A+n/2, lda, A+n/2+n/2*lda, lda, hwork, syrk_nb);

//     tc_rpotrf(cublas_handle, cusolver_handle, n/2, A+n/2+n/2*lda, lda, work, hwork, nb, devInfo);

//     return;
// }


int rec_cholesky(hipblasHandle_t cublas_handle,hipsolverHandle_t cusolver_handle, double *A, long ldA,int n, int *devInfo)
{
    // 0.判断是否需要调用cuSolver
    if(n <= 64)
    {
        int Lwork;
        
        hipsolverDnDpotrf_bufferSize( cusolver_handle,
                 HIPBLAS_FILL_MODE_LOWER,
                 n,
                 A,
                 ldA,
                 &Lwork);

        double *work;
        hipMalloc((void**)&work, sizeof(double)*Lwork);

        hipsolverDnDpotrf(cusolver_handle, HIPBLAS_FILL_MODE_LOWER, 
        n, A, ldA, work, Lwork, devInfo);

        hipFree(work);

        return 0;

    }
    
    // 1.把A分解为A11、A12、A21、A22
    int n2 = n/2;

    // 2.对A11进行Cholesky分解,求出L11
    double *A11 = A;
    int ldA11 = ldA;
    rec_cholesky(cublas_handle,cusolver_handle, A11, ldA11, n2, devInfo);

    double dOne = 1.0;

    // 3.对A21进行trsm,求出L21 = L11^-1 * A21
    double *A21 = A + n2;
    cublasDtrsm(cublas_handle,
                            HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
                            HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT,
                           n2, n2,
                           &dOne,
                           A11, ldA,
                           A21, ldA);

    // 4.对A22 - L21 * L21^T进行syrk
    double dNegOne = -1.0;
    double *A22 = A + n2 + n2*ldA;
    cublasDsyrk(cublas_handle,
                            HIPBLAS_FILL_MODE_LOWER,
                            HIPBLAS_OP_N,
                            n2, n2,
                            &dNegOne,
                            A21, ldA,
                            &dOne,
                            A22, ldA);

    // 5.递归调用此函数，对A22 - L21 * L21^T进行Cholesky分解
    rec_cholesky(cublas_handle, cusolver_handle, A22, ldA, n2, devInfo);

    return 0;
}


int main(int argc, char *argv[])
{
    if (2 != argc)
    {
        cout << "Usage(b = nb in ZY): AppName <n>" << endl;
        return 0;
    }

    int m, n;
    m = n = atol(argv[1]);


    // 1.初始化cuBLAS和cuSolver
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    hipsolverHandle_t cusolver_handle;
    hipsolverDnCreate(&cusolver_handle);

    // 2.初始化矩阵A
    // int n = 1024;
    double *A;
    
    hipMalloc((void**)&A, sizeof(double)*n*n);

    // generateUniformMatrix(A, n, n);
    // dim3 gridDim((m + 31) / 32, (n + 31) / 32);
    // dim3 blockDim(32, 32);
    // launchKernel_CpyMatrixL2U(gridDim, blockDim, n, A, n);

    // 3.调用递归函数rec_cholesky
    int devInfo;
    hipMalloc((void**)&devInfo, sizeof(int));

    rec_cholesky(cublas_handle, cusolver_handle, A, n, n, &devInfo);

    // 4.释放cuBLAS和cuSolver
    hipblasDestroy(cublas_handle);
    hipsolverDnDestroy(cusolver_handle);

    // 5.释放矩阵A
    hipFree(A);

    return 0;
}

